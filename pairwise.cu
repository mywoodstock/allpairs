#include "hip/hip_runtime.h"
/***
 *  $Id$
 **
 *  File: example-cuda.cu
 *  Created: Apr 17, 2010
 *
 *  Author: Abhinav Sarje <abhinav.sarje@gmail.com>
 */

#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <iomanip>

#include "io4example.hpp"

/**
 * kernel functions for the cuda device
 */

/**
 * lp norm
 */

// this kernel computes only partial lpnorm result for a slice
template<typename value_type>
__device__
value_type lpnorm_slice(const value_type* a, const value_type* b, unsigned int d, int p) {
	value_type temp = 0;

	for(unsigned int k = 0; k < d; ++ k) {
		temp += pow(fabs(a[k] - b[k]), p);
	} // for

	return temp;
} // lpnorm_slice()

// after all slices are computed into v, this computes the final value in lpnorm
template<typename value_type>
__device__
value_type lpnorm_acc(const value_type v, value_type inv_p) {
	return pow(v, inv_p);
} // lpnorm_acc()

/**
 * dot product
 */

template<typename value_type>
__device__
value_type dot_product(const value_type* a, const value_type* b, unsigned int d) {
	value_type temp = 0;
	for(unsigned int k = 0; k < d; ++ k) {
		temp += a[k] * b[k];
	} // for

	return temp;
} // dot_product()


/**
  * kernel function for device
  */

template<typename value_type>
__global__
void process_nonsym(unsigned int n, unsigned int d, int p, value_type inv_p,
		unsigned int ds, unsigned int num_subtiles,
		value_type *M, value_type *D) {
	// a block is block_rows x tile_cols, and is responsible to
	// compute a tile of size tile_rows x tile_cols
	// (following the weird y x x convention)
	// assuming that D is initialized to 0

	const int block_rows = blockDim.y;
	const int tile_cols = blockDim.x;
	const int tile_rows = block_rows * num_subtiles;

	// number of slices
	const int num_slices = ceil((float)d / ds );
	unsigned int ds_last = d - ds * (num_slices - 1);

	unsigned int a_idx = tile_cols * blockIdx.x + threadIdx.x;
	unsigned int b_idx = tile_rows * blockIdx.y + threadIdx.y;

	// shared memory
	extern __shared__ value_type shared_M[];

	// shared memory for x and y vectors
	value_type *shared_x = shared_M;
	value_type *shared_y = shared_M + tile_cols * ds;
	value_type *a, *b;

	value_type *M_slice = M;

	unsigned int shared_a_idx = ds * threadIdx.x;
	unsigned int shared_b_idx = ds * threadIdx.y;

	for(int slice = 0; slice < num_slices; ++ slice) {
		unsigned int ds_curr = ds;
		if(slice == num_slices - 1) ds_curr = ds_last;

		// first get the x vectors into shared mem
		// these will be reused for the whole tile
		int num_transfers = ceil((float)ds_curr / block_rows);

		__syncthreads();
		for(int i = 0; i < num_transfers; ++i) {
			unsigned int d_idx = block_rows * i + threadIdx.y;
			if(d_idx < ds_curr && threadIdx.x < tile_cols)
				shared_x[ds * threadIdx.x + d_idx] = M_slice[d * a_idx + d_idx];
		} // for
		__syncthreads();

		// get the x vector slice
		if(a_idx < n) a = shared_x + shared_a_idx;

		value_type *M_ptr = M_slice + (tile_rows * d * blockIdx.y);
		num_transfers = ceil((float)ds_curr / tile_cols);

		// for all subtiles - this brings in the sequential part to reuse x vectors
		for(unsigned int subtile = 0; subtile < num_subtiles; ++ subtile) {

			// fill the shared memory with the subtile y vectors
			__syncthreads();	// to make ensure completion of last iteration before moving memory
			for(int i = 0; i < num_transfers; ++ i) {
				unsigned int d_idx = tile_cols * i + threadIdx.x;
				if((d_idx < ds_curr) && (threadIdx.y < block_rows) &&
						(b_idx + block_rows * subtile < n)) {
					shared_y[ds * threadIdx.y + d_idx] = M_ptr[d * threadIdx.y + d_idx];
				} // if
			} // for
			__syncthreads();	// wait for all threads to finish moving memory

			if((a_idx < n) && (b_idx + block_rows * subtile < n)) {
				// now get the y vector from shared_y
				b = shared_y + shared_b_idx;

				// perform computation
				unsigned int D_idx = n * (b_idx + block_rows * subtile) + a_idx;
#ifdef MATMUL
				D[D_idx] += dot_product(a, b, ds_curr);
#else
				D[D_idx] += lpnorm_slice(a, b, ds_curr, p);
#endif
			} // if

			M_ptr += block_rows * d;
		} // for subtile

		M_slice += ds_curr;
	} // for slice

#ifdef MATMUL
#else
	// compute the final values after accumulation
	for(unsigned int subtile = 0; subtile < num_subtiles; ++ subtile) {
		if(a_idx < n && b_idx < n) {
			// perform aggregation computation
			unsigned int D_idx = n * (b_idx + block_rows * subtile) + a_idx;
			if(a_idx < n && b_idx + block_rows * subtile < n) {
				D[D_idx] = lpnorm_acc(D[D_idx], inv_p);
			} // if
		} // if
	} // for
#endif
} // process_nonsym()


// kernel for cuda device
template<typename value_type>
__global__
void process_sym(int p, unsigned int n, unsigned int d, value_type* M, value_type inv_p, value_type *D) {
} // process_sym()


/**
  * kernel for host
  */

template <typename value_type>
__host__
void process_nonsym_host(int p, unsigned int n, unsigned int d, value_type* M, value_type inv_p, value_type* D) {

    value_type* D_ptr = D;
    value_type* x = M;
    for (unsigned int i = 0; i < n; ++i) {
        value_type* y = M;
        for (unsigned int j = 0; j < n; ++j) {
            for (unsigned int k = 0; k < d; ++k) {
                D_ptr[j] += pow(fabs(x[k] - y[k]), p);
            }
			D_ptr[j] = pow(D_ptr[j], inv_p);
            y += d;
        } // y
        D_ptr += n;
        x += d;
    } // x
} // process_nonsym


/**
 * Miscellaneous stuff
 */

void device_information() {
	int device_count;
	hipGetDeviceCount(&device_count);

	for(int i = 0; i < device_count; ++ i) {
		hipDeviceProp_t device_prop;
		hipGetDeviceProperties(&device_prop, i);

		if(i == 0) {
			if(device_prop.major == 9999 && device_prop.minor == 9999)
				std::cout << "There is no device supporting CUDA." << std::endl;
			else if(device_count == 1)
				std::cout << "There is 1 device supporting CUDA" << std::endl;
			else std::cout << "There are " << device_count << " devices supporting CUDA" << std::endl;
		}
	} // for
} // device_information()


/**
  * Main stuff
  */

int main(int argc, char* argv[]) {
    if (argc != 11) {
        std::cout << "Usage: " << argv[0]
			<< " n d p infile outfile sym block_c block_r subtiles slice"
			<< std::endl;
        return 0;
    } // if

    int n = atoi(argv[1]);
    int d = atoi(argv[2]);
    int p = atoi(argv[3]);	// this does not matter for matrix multiplication
    bool sym = atoi(argv[6]);

	const unsigned int BLOCK_SIZE_C = atoi(argv[7]);
	const unsigned int BLOCK_SIZE_R = atoi(argv[8]);
	const int NUM_SUBTILE = atoi(argv[9]);
	const unsigned int ds = atoi(argv[10]);

    typedef double value_type;

	//size_t free, total;
	//hipMemGetInfo(&free, &total);
	//std::cout << "Free: " << (float)free/(1024*1024*1024)
	//		<< ", Total: " << (float)total/(1024*1024*1024)
	//		<< std::endl;
	//device_information();

    unsigned int out_size = n * n;
    if (sym == true) out_size = ((out_size - n) >> 1);

	// use page-locked host memory
	value_type* M;
	if(hipHostAlloc((void **) &M, n * d * sizeof(value_type), 0) != hipSuccess) {
		std::cerr << "Error: cannot allocate page-locked memory." << std::endl;
		return -1;
	} // if
	value_type* D;
	if(hipHostAlloc((void **) &D, out_size * sizeof(value_type), hipHostMallocWriteCombined)
			!= hipSuccess) {
		std::cerr << "Error: cannot allocate page-locked memory." << std::endl;
		hipHostFree(M);
		return -1;
	} // if
	// set output to 0
	if(memset(D, 0, out_size * sizeof(value_type)) == NULL) {
		std::cerr << "Error: memset failed." << std::endl;
		hipHostFree(D);
		hipHostFree(M);
		return -1;
	} // if

	// allocate device memories
	value_type* d_M;
	value_type* d_D;
	if(hipMalloc((void **) &d_M, n * d * sizeof(value_type)) != hipSuccess) {
		std::cerr << "Error: cannot allocate device memory." << std::endl;
		hipHostFree(D);
		hipHostFree(M);
		return -1;
	} // if
	if(hipMalloc((void **) &d_D, out_size * sizeof(value_type)) != hipSuccess) {
		std::cerr << "Error: cannot allocate device memory." << std::endl;
		hipFree(d_M);
		hipHostFree(D);
		hipHostFree(M);
		return -1;
	} // if

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsedTime = 0.0;

    value_type inv_p = (value_type)1.0 / p;

    // load M from file
    if (load_M_row(argv[4], n, d, M) == false) {
        std::cerr << "Error: can't read file " << argv[4] << "." << std::endl;
		hipFree(d_D);
		hipFree(d_M);
		hipHostFree(D);
		hipHostFree(M);
        return -1;
    } // if

	hipEventRecord(start, 0);

	// copy M from host to device
	// and set device output memory to 0
	hipMemcpy(d_M, M, n * d * sizeof(value_type), hipMemcpyHostToDevice);
	hipMemset(d_D, 0, out_size * sizeof(value_type));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	std::cout << "[device memory time: " << elapsedTime << " ms]" << std::endl;

    std::cout << "* processing ..." << std::endl;

	hipEventRecord(start, 0);

	// set grid and block sizes
	//dim3 block(i/j, j);
	const unsigned int c = BLOCK_SIZE_C;
	const unsigned int r = BLOCK_SIZE_R * NUM_SUBTILE;

	dim3 block(BLOCK_SIZE_C, BLOCK_SIZE_R);
	dim3 grid(ceil((float)n / block.x), ceil((float)n / r));
	size_t s_M_size = (BLOCK_SIZE_R + c) * ds * sizeof(value_type);

	// this is just for this case, this may change when the kernel is changed!
	unsigned int num_reg = block.x * block.y;
	if(sizeof(value_type) == 4) num_reg *= 36;
	else num_reg *= 48;

	std::cout << "-------------------------------" << std::endl;
	std::cout << "+ grid:  " << grid.y << "x" << grid.x << std::endl;
	std::cout << "+ block: " << block.y << "x" << block.x << std::endl;
	std::cout << "+ tile:  " << r << "x" << c << std::endl;
	std::cout << "+ subtiles: " << NUM_SUBTILE << std::endl;
	std::cout << "+ dynamic shared memory: " << s_M_size << " B" << std::endl;
	std::cout << "+ registers per block: " << num_reg << std::endl;
	std::cout << "-------------------------------" << std::endl;

	// invoke the kernel function
   	std::cout << "* on device" << std::endl;
   	if(sym == false) {
		process_nonsym<<< grid, block, s_M_size >>>(n, d, p, inv_p, ds, NUM_SUBTILE, d_M, d_D);
	} else process_sym<<< grid, block >>>(p, n, d, d_M, inv_p, d_D);

	// check for any errors in kernel launch
	hipError_t error_code = hipGetLastError();
	if(error_code != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(error_code) << std::endl;
		hipFree(d_D);
		hipFree(d_M);
		hipHostFree(D);
		hipHostFree(M);
		return -1;
	} // if

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	std::cout << "[device compute time: " << elapsedTime << " ms]" << std::endl;

	hipEventRecord(start, 0);

	// trasnfer results (D) from device memory to host memory
	hipMemcpy(D, d_D, out_size * sizeof(value_type), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	std::cout << "[device memory time: " << elapsedTime << " ms]" << std::endl;

	/*hipEventRecord(start, 0);
    std::cout << "* on host" << std::endl;
	process_nonsym_host(p, n, d, M, inv_p, D);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	std::cout << "Host Time: " << elapsedTime << "ms" << std::endl;
	*/

    bool res = false;
    if (sym == false) res = store_D_nonsym(argv[5], D, n);
    else res = store_D_sym(argv[5], D, n);

    if (res == false) {
        std::cerr << "Error: can't write file " << argv[5] << std::endl;
		hipFree(d_D);
		hipFree(d_M);
		hipHostFree(D);
		hipHostFree(M);
        return -1;
    } // if

	// free the events
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// free host and device memories
	hipFree(d_D);
	hipFree(d_M);
	hipHostFree(D);
	hipHostFree(M);

    return 0;
} // main
